#include "hip/hip_runtime.h"
    #include <iostream>
    #include <algorithm>
    #include <sys/time.h>
    #include "hip/hip_runtime.h"
    #include "hip/hip_runtime.h"
    using namespace std;

    #define RADIUS 3

    __global__ void stencil_1D(int *in, int *out, int N) {
        __shared__ int temp[blockDim.x + 2*RADIUS];
        int gindex = threadIdx.x + blockIdx.x*blockDim.x;
        int lindex = threadIdx.x + RADIUS;

        if (threadIdx.x < RADIUS) {
            if (gindex < RADIUS) {
                temp[lindex - RADIUS] = 0;
            }
            else {
                temp[lindex - RADIUS] = in[gindex - RADIUS];
            }
            if (gindex + blockDim.x < N) {
                temp[lindex + blockDim.x] = in[gindex + BLOCK_SIZE];
            }
            else {
                temp[lindex + blockDim.x] = 0;
            }
        }

        if (gindex < N) {
            temp[lindex] = in[gindex];
        }

        __syncthreads();

        if (gindex < N) {
            int result = 0;
            for (int offset=-RADIUS; offset<=RADIUS; offset++) {
                result += temp[lindex + offset];
            }

            out[gindex] = result;
        }
    }


    void fill_ints(int *x, int n) {
        fill_n(x, n, 1);
    }

    int main(int argc, char *argv[]) {
        int *h_in, *h_out;
        int *d_in, *d_out;
        int N, size, b_size;
        struct timeval t1, t2;
        double t_total;

        if (argc < 2) {
            printf("Error: you must indicate the length of the array\n");
            return 1;
        }

        if (argc < 3) {
            printf("Error: you must indicate the block size\n");
            return 1;
        }

        N = atoi(argv[1]);
        size = N * sizeof(int);
        b_size = atoi(argv[2]);

        h_in = (int*) malloc(size);
        h_out = (int*) malloc(size);
        fill_ints(h_in, N);
        fill_ints(h_out, N);

        hipMalloc((void**) &d_in, size);
        hipMalloc((void**) &d_out, size);

        gettimeofday(&t1, NULL);
        hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);
        hipMemcpy(d_out, h_out, size, hipMemcpyHostToDevice);

        stencil_1D<<<(N+b_size-1)/b_size,b_size>>>(d_in, d_out, N);

        hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);

        gettimeofday(&t2, NULL);

        /*
        printf("Output: \n");
        for (int i=0; i<N; i++) {
            printf("%d ", h_out[i]);
        }
        printf("\n");
        */

        t_total = (t2.tv_sec - t1.tv_sec)*1000000.0 + (t2.tv_usec - t1.tv_usec);
        printf("%d\t%f\n", N, t_total);

        free(h_in);
        free(h_out);
        hipFree(d_in);
        hipFree(d_out);
        return 0;
    }
